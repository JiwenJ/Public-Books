#include "hip/hip_runtime.h"
/*
 * @brief The fifth CUDA quadratic sum program with reduction tree.
 * @author Deyuan Qiu
 * @date June 22nd, 2009
 * @file gpu_quadratic_sum_5.cu
 */

#include <iostream>
#include "cutil.h"

#define DATA_SIZE 1048576	//data of 4 MB
#define BLOCK_NUM	32
#define THREAD_NUM	256

using namespace std;

void GenerateNumbers(int *number, int size)
{
    for(int i = 0; i < size; i++)	number[i] = rand() % 10;
}

//The kernel implemented by a global function: called from host, executed in device.
__global__ static void sumOfSquares(int *num, int* result,
    clock_t* time)
{
    extern __shared__ int shared[];
    const int tid = threadIdx.x;
    const int bid = blockIdx.x;
    int i;
    int offset = 1;
    if(tid == 0) time[bid] = clock();
    shared[tid] = 0;
    for(i = bid * THREAD_NUM + tid; i < DATA_SIZE;
        i += BLOCK_NUM * THREAD_NUM) {
       shared[tid] += num[i] * num[i];
    }

    __syncthreads();
    offset = THREAD_NUM / 2;
	while (offset > 0) {
		if (tid < offset) {
			shared[tid] += shared[tid + offset];
		}
		offset >>= 1;
		__syncthreads();
	}

	if (tid == 0) {
		result[bid] = shared[0];
		time[bid + BLOCK_NUM] = clock();
    }
}

int main(int argc, char **argv)
{
	CUT_DEVICE_INIT(argc, argv);

	//allocate host page-locked memory
	int *data, *sum;
	CUDA_SAFE_CALL(hipHostMalloc((void**)&data, DATA_SIZE*sizeof(int)));
	GenerateNumbers(data, DATA_SIZE);
	CUDA_SAFE_CALL(hipHostMalloc((void**)&sum, BLOCK_NUM*sizeof(int)));
	clock_t *time_used;
	CUDA_SAFE_CALL(hipHostMalloc((void**)&time_used, sizeof(clock_t) * BLOCK_NUM * 2));

	//allocate device memory
	int *gpudata, *result;
	clock_t *time;
	CUDA_SAFE_CALL(hipMalloc((void**) &gpudata, sizeof(int) * DATA_SIZE));
	CUDA_SAFE_CALL(hipMalloc((void**) &result, sizeof(int) * BLOCK_NUM));
	CUDA_SAFE_CALL(hipMalloc((void**) &time, sizeof(clock_t) * BLOCK_NUM * 2));
	CUDA_SAFE_CALL(hipMemcpy(gpudata, data, sizeof(int) * DATA_SIZE, hipMemcpyHostToDevice));

	//Using THREAD_NUM scalar processer and shared memory.
	sumOfSquares<<<BLOCK_NUM, THREAD_NUM, THREAD_NUM * sizeof(int)>>>(gpudata, result, time);

	CUDA_SAFE_CALL(hipMemcpy(sum, result, sizeof(int) * BLOCK_NUM, hipMemcpyDeviceToHost));
	CUDA_SAFE_CALL(hipMemcpy(time_used, time, sizeof(clock_t) * BLOCK_NUM * 2, hipMemcpyDeviceToHost));

	//sum up on CPU
	int final_sum = 0;
	for (int i = 0; i < BLOCK_NUM; i++)	final_sum += sum[i];

	//calculate the time: minimum start time - maximum end time.
	clock_t min_start, max_end;
	min_start = time_used[0];
	max_end = time_used[BLOCK_NUM];
	for (int i = 1; i < BLOCK_NUM; i++) {
		if (min_start > time_used[i])
			min_start = time_used[i];
		if (max_end < time_used[i + BLOCK_NUM])
			max_end = time_used[i + BLOCK_NUM];
	}

	printf("sum: %d  time: %d\n", final_sum, max_end - min_start);

	//Clean up
	CUDA_SAFE_CALL(hipFree(time));
	CUDA_SAFE_CALL(hipFree(result));
	CUDA_SAFE_CALL(hipFree(gpudata));
	CUDA_SAFE_CALL(hipHostFree(sum));
	CUDA_SAFE_CALL(hipHostFree(data));
	CUDA_SAFE_CALL(hipHostFree(time_used));

	return EXIT_SUCCESS;
}
