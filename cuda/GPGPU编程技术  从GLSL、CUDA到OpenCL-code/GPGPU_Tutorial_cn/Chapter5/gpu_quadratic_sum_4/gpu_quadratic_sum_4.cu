#include "hip/hip_runtime.h"
/*
 * @brief The forth CUDA quadratic sum program with increased threads.
 * @author Deyuan Qiu
 * @date June 21st, 2009
 * @file gpu_quadratic_sum_4.cu
 */

#include <iostream>
#include "cutil.h"

#define DATA_SIZE 1048576	//data of 4 MB
#define BLOCK_NUM	32
#define THREAD_NUM	256

using namespace std;

void GenerateNumbers(int *number, int size)
{
    for(int i = 0; i < size; i++)	number[i] = rand() % 10;
}

//The kernel implemented by a global function: called from host, executed in device.
__global__ static void sumOfSquares(int *num, int* result,
    clock_t* time)
{
    const int tid = threadIdx.x;
    const int bid = blockIdx.x;
    int sum = 0;
    int i;
    if(tid == 0) time[bid] = clock();
    for(i = bid * THREAD_NUM + tid; i < DATA_SIZE;
        i += BLOCK_NUM * THREAD_NUM) {
       sum += num[i] * num[i];
    }

    result[bid * THREAD_NUM + tid] = sum;
    if(tid == 0) time[bid + BLOCK_NUM] = clock();
}

int main(int argc, char **argv)
{
	CUT_DEVICE_INIT(argc, argv);

	//allocate host page-locked memory
	int *data, *sum;
	CUDA_SAFE_CALL(hipHostMalloc((void**)&data, DATA_SIZE*sizeof(int)));
	GenerateNumbers(data, DATA_SIZE);
	CUDA_SAFE_CALL(hipHostMalloc((void**)&sum, BLOCK_NUM*THREAD_NUM*sizeof(int)));
	clock_t *time_used;
	CUDA_SAFE_CALL(hipHostMalloc((void**)&time_used, sizeof(clock_t) * BLOCK_NUM * 2));

	//allocate device memory
	int *gpudata, *result;
	clock_t *time;
	CUDA_SAFE_CALL(hipMalloc((void**) &gpudata, sizeof(int) * DATA_SIZE));
	CUDA_SAFE_CALL(hipMalloc((void**) &result, sizeof(int) * THREAD_NUM * BLOCK_NUM));
	CUDA_SAFE_CALL(hipMalloc((void**) &time, sizeof(clock_t) * BLOCK_NUM * 2));
	CUDA_SAFE_CALL(hipMemcpy(gpudata, data, sizeof(int) * DATA_SIZE, hipMemcpyHostToDevice));

	//Using THREAD_NUM scalar processer.
	sumOfSquares<<<BLOCK_NUM, THREAD_NUM, 0>>>(gpudata, result, time);

	CUDA_SAFE_CALL(hipMemcpy(sum, result, sizeof(int) * THREAD_NUM * BLOCK_NUM, hipMemcpyDeviceToHost));
	CUDA_SAFE_CALL(hipMemcpy(time_used, time, sizeof(clock_t) * BLOCK_NUM * 2, hipMemcpyDeviceToHost));

	//sum up on CPU
	int final_sum = 0;
	for (int i = 0; i < THREAD_NUM * BLOCK_NUM; i++)	final_sum += sum[i];

	//calculate the time: minimum start time - maximum end time.
	clock_t min_start, max_end;
	min_start = time_used[0];
	max_end = time_used[BLOCK_NUM];
	for (int i = 1; i < BLOCK_NUM; i++) {
		if (min_start > time_used[i])
			min_start = time_used[i];
		if (max_end < time_used[i + BLOCK_NUM])
			max_end = time_used[i + BLOCK_NUM];
	}

	printf("sum: %d  time: %d\n", final_sum, max_end - min_start);

	//Clean up
	CUDA_SAFE_CALL(hipFree(time));
	CUDA_SAFE_CALL(hipFree(result));
	CUDA_SAFE_CALL(hipFree(gpudata));
	CUDA_SAFE_CALL(hipHostFree(sum));
	CUDA_SAFE_CALL(hipHostFree(data));
	CUDA_SAFE_CALL(hipHostFree(time_used));

	return EXIT_SUCCESS;
}
